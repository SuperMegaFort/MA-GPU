#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"
#include "Limits.h"

/*----------------------------------------------------------------------*\
 |*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*--------------------------*\
 |*	private		     *|
 \*-------------------------*/

#include "losetime.cu.h"

static __device__ void processS(int* ptrGmSlice1 , int* ptrDevV2 , int* ptrGmSliceW , int sLocalSlice);

/*----------------------------------------------------------------------*\
 |*			Implementation					*|
 \*---------------------------------------------------------------------*/

/**
 * <pre>
 * Inputs:
 * 	- ptrGmSlice1 : pointeur sur la slice du vecteur1
 * 	- ptrGmSlice2 : pointeur sur la slice du vecteur2
 * 	- ptrGmSlicew : pointeur sur la slice du vecteur resultat w
 *
 * 	- n_by_slice  : le nombre d'element dans une slice
 * 	- sid	      : l'index de slice in [0,nbSlice[		(Pas used ici)
 *
 *
 * Note:
 * 	Ce meme kernel sera utiliser pour nos 3 versions:
 * 		- baseline (sans stream, ou stream unique)
 * 		- bistream (bislice)
 * 		- tristram
 *
 * 	Version baseline (sans stream)
 * 		- ptrGmSlice1 = ptrGMV1	  pointeur sur le debut du vecteur complet v1
 * 		- n_by_slice  = n 	  le nombre d'element total du vecteur
 * 		- sid         = 0	  unique slice, la slice est le vecteur complet
 *
 *   	Version bistream (bislice)
 * 		- ptrGmSlice1 = 	  pointeur sur le debut d'une des slices du vecteur v1
 * 		- n_by_slice  = 	  le nombre d'element d'une slice
 * 		- sid         = 0 ou 1	  selon si on travailler la slice 0 ou la slice 1
 *
 *   	Version tristream
 * 		- ptrGmSlice1 = ptrGMV1	   pointeur sur le debut d'une des slices du vecteur v1
 * 		- n_by_slice  = n 	   le nombre d'element d'une slice
 * 		- sid      		   le slice index

 * </pre>
 */
__global__ void addVector(int* ptrGmSlice1 , int* ptrGmSlice2 , int* ptrGmSliceW , int n_by_slice , int sid = 0)
    {
    //  Indications:
    //	   (I1) Entrelacement sur le slice et uniquement sle slice
    //	   (I2) Aidez vous ensuite de la fonction secondaire processS ci-dessous

    // TODO stream

    // int sGlobal = sLocalSlice + (sid * n_by_slice); // global au vecteur en partant du debut, pas used ici, car on a deja un pointeur sur le debut de la slice
    }

/*--------------------------*\
 |*	private		     *|
 \*-------------------------*/

/**
 * calcul le resultat de la case sLocalSlice in [0,n_by_slice[ de la slice "courante"
 */
__device__ void processS(int* ptrGmSlice1 , int* ptrGmSlice2 , int* ptrGmSliceW , int sLocalSlice)
    {
    // Indications:
    //	(1) additioner la composante s:	u=v1(s)+v2(s)  (avec s= LocalSlice)
    //	(2) appeler loseTime sur u, loseTime ne modifie pas u,loseTime est un fonction identity
    //	(3) le resultat final est loseTime(u), ie la cases s additionner, mais avec une perte de temps
    //
    //	 		resultatS=loseTime(u)

    // TODO stream

    // TIPS : pour debuguer, mettez au début:
    //
    //		ptrGmSliceW[s]=sGlobal;
    //
    //	Si vous avez un bug, ca permetra de savoir si ca vient du host ou du device

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

