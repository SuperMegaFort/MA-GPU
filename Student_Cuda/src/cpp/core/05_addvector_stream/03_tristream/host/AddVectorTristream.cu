#include "hip/hip_runtime.h"
#include "AddVectorTristream.h"

#include <iostream>
#include <assert.h>

#include "Kernel.h"
#include "GM.h"
#include "Bandwidth.h"
#include "VectorTools.h"
#include "Stream.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addVector(int* ptrGmSlice1 , int* ptrGmSlice2 , int* ptrGmSliceW , int n_by_slice , int sid = 0);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVectorTristream::AddVectorTristream(const Grid& grid , int* ptrV1 , int* ptrV2 , int* ptrW , int n , int nbSlice , bool isVerbose) :
	RunnableGPU(grid, title(nbSlice) + "-" + to_string(n), isVerbose), // classe parente
	//
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n), //
	nbSlice(nbSlice)
    {
    assert(n % nbSlice == 0);
    assert(nbSlice >= 3);

    this->sizeVector = n * sizeof(int); // octet
    this->SIZE_SLICE = sizeVector / nbSlice;
    this->N_BY_SLICE = n / nbSlice;

    // MM (malloc Device)
	{
	GM::malloc(&ptrGMV1, sizeVector);
	GM::malloc(&ptrGMV2, sizeVector);
	GM::malloc(&ptrGMW, sizeVector);
	}

    // Stream
	{
	// TODO stream, see attribute in .h
	assert(false); // to remove once coded
	}
    }

AddVectorTristream::~AddVectorTristream(void)
    {
    //MM (device free)
	{
	GM::free(ptrGMV1);
	GM::free(ptrGMV2);
	GM::free(ptrGMW);
	}

    // Stream
	{
	// TODO stream, see attribute in .h
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * Conseils :
 *
 * 	(C1) Commencer par les warmup pour un nombre de slice petit et fixe, puis passer seulement ensuite a la generalisation.
 *
 * 	(C2) Pour les warmup, prenez une taille n de vecteur petit (pour pouvoir afficher le resultat)
 * 	     Aller dans:
 *
 * 	    		VectorTools::n(); // activez la ligne debug provisoirement, et dans nDebug() prenez une valeur petite et divisible par votre nombre se slice
 *
 * 	(C3) Travailler au début avec mainUse.cpp (utiliser le blog debug fournit avec flag de verbosite a false avec !)
 */
void AddVectorTristream::run()
    {
    // warmup
	{
	run3Slice();
	//run4Slice();
	//run5Slice();
	}

    //runGeneric(); // TODO stream a activer une fois le warmup valider

    // Warning : le mode LaunchModeMOO::TEST dans main a besoin du code generic coder et activer

    // synchronise
	{
	// TODO stream (attendre la fin de toutes les stream de ce tp)
	}
    }

/*--------------------------------------*\
 |*	Differentes Versions		*|
 \*-------------------------------------*/

// pour des raisons de clareter, le code se trouve dans les .h includer ci-dessous (il se trouve dans le folder helper)

#include "run3Slice.h"
#include "run4Slice.h"
#include "run5Slice.h"
#include "runGeneric.h"

/*--------------------------------------*\
 |*		Tools			*|
 \*-------------------------------------*/

/**
 * sid=sliceIndex in [0,nbSlice[
 * return decalage a effectuer pour pointer sur le premier element d'une slice.
 *
 * Exemple:
 *
 * 	Si les slices ont 4 cases, et qu'il a 3 slices:
 *
 * 	sid=0  offsetSlice(0) vaut 0	ptrGMV1+offsetSlice(0) pointe sur la premiere case du slice 0   (de v1)
 * 	sid=1  offsetSlice(1) vaut 4	ptrGMV1+offsetSlice(1) pointe sur la premiere case du slice 1	(de v1)
 * 	sid=2  offsetSlice(2) vaut 8	ptrGMV1+offsetSlice(2) pointe sur la premiere case du slice 2	(de v1)
 */
int AddVectorTristream::offsetSlice(int sid)
    {
    assert(false); // remove once coded
    return 0; // TODO stream
    }

/**
 * copyHtoD: la slice sid pour
 *
 * 	v1
 * 	v2
 * ou
 * 	sid=sliceIndex in [0,nbSlice[
 */
void AddVectorTristream::copyHtoD(int sid , hipStream_t stream)
    {
    const int OFFSET_SLICE = offsetSlice(sid);

    // Indication:
    //		pour le slice de v1
    //		pour le slice de v2

    // TODO stream
    assert(false); // remove once coded
    }

/**
 * copyDtoH: la slice sid pour
 *
 * 	w
 * ou
 * 	sid=liceIndex in [0,nbSlice[
 */
void AddVectorTristream::copyDtoH(int sid , hipStream_t stream)
    {
    const int OFFSET_SLICE = offsetSlice(sid);

    // TODO stream
    assert(false); // remove once coded
    }

/**
 * lance le kernel de calcul pour la slice sid
 * ou
 * 	sid=sliceIndex in [0,nbSlice[
 */
void AddVectorTristream::kernelSlice(int sid , hipStream_t stream)
    {
    const int OFFSET_SLICE = offsetSlice(sid);

    // TODO stream
    assert(false); // remove once coded
    }

/*------------------------*\
|*  	secondaire  	   *|
 \*-----------------------*/

/**
 * override
 */
double AddVectorTristream::getInputGO()
    {
    return ((long)2 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/**
 * override
 */
double AddVectorTristream::getOutputGO()
    {
    return ((long)1 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

string AddVectorTristream::title(int nbSlice)
    {
    if (VectorTools::isDMA())
	{
	return "Addvector-tristream-slice" + to_string(nbSlice) + "-DMA-int";
	}
    else
	{
	return "Addvector-tristream-slice" + to_string(nbSlice) + "-DMA-int";
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
