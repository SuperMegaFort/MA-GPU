#include "hip/hip_runtime.h"
#include "AddVector.h"

#include <iostream>
#include <assert.h>

#include "Kernel.h"
#include "GM.h"
#include "Bandwidth.h"
#include "VectorTools.h"
#include "Stream.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addVector(int* ptrGmSlice1 , int* ptrGmSlice2 , int* ptrGmSliceW , int n_by_slice , int sid = 0);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVector::AddVector(const Grid& grid , int* ptrV1 , int* ptrV2 , int* ptrW , int n , bool isVerbose) :
	RunnableGPU(grid, title() + "-" + to_string(n), isVerbose), // classe parente
	//
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n)
    {
    this->sizeVector = n * sizeof(int); // octet

    // MM (malloc Device)
	{
	GM::malloc(&ptrGMV1, sizeVector);
	GM::malloc(&ptrGMV2, sizeVector);
	GM::malloc(&ptrGMW, sizeVector);
	}

    }

AddVector::~AddVector(void)
    {
    //MM (device free)
	{
	GM::free(ptrGMV1);
	GM::free(ptrGMV2);
	GM::free(ptrGMW);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * override
 */
void AddVector::run()
    {
    // MM (copy Host->Device)
	{
	Bandwidth bandwidth(sizeVector * 2, "[" + title() + "] : Host -> GM :");

	GM::memcpyHToD(ptrGMV1, ptrV1, sizeVector);
	GM::memcpyHToD(ptrGMV2, ptrV2, sizeVector);

	if (isVerbose) // dans Runable ou RunnableGPU
	    {
	    cout << bandwidth << endl;
	    }
	}

    // call kernel
	{
    addVector<<<dg,db>>>(ptrGMV1, ptrGMV2, ptrGMW, n);	// assynchrone
    }

// MM (Device -> Host)
    {
    Bandwidth bandwidth(sizeVector, "[" + title() + "] : GM -> Host :");

    GM::memcpyDToH(ptrW, ptrGMW, sizeVector);

    if (isVerbose)
	{
	cout << bandwidth << endl;
	}
    }
}

/**
 * override
 */
double AddVector::getInputGO()
{
return ((long)2 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
}

/**
 * override
 */
double AddVector::getOutputGO()
{
return ((long)1 * (long)n * (long)sizeof(half)) / (double)((long)1024 * (long)1024 * (long)1024);
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

string AddVector::title()
{
if (VectorTools::isDMA())
    {
    return "Addvector-DMA-int";
    }
else
    {
    return "Addvector-int";
    }
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
