#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * <pre>
 * Output :
 *
 * 		void required, car kernel is asynchrone!
 *
 * Contrainte:
 * 		pattern 1<-->1
 * 		On associe un thread a chaque case du tableau
 * </pre>
 */
__global__ void addArray11(float* ptrGMV1 , float* ptrGMV2 , float* ptrGMW , int n)
    {
    const int TID = Thread2D::tid();

    // TODO addArray11
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

