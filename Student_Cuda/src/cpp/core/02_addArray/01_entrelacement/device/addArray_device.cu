#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * output : void required, car kernel is asynchrone
 * pattern entrelacement
 */
__global__ void addArray(float* ptrGMV1 , float* ptrGMV2 , float* ptrGMW , int n)
    {
    const int NB_THREAD = Thread2D::nbThread();
    const int TID = Thread2D::tid();
    int s = TID;

    while(s < n){
	ptrGMW[s] = ptrGMV1[s] + ptrGMV2[s];
	s += NB_THREAD;

    }

    // Debug, facultatif
//    if (TID == 0)
//	{
//	printf("Coucou from device tid = %d", TID);
//	}

    // TODO addArray
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

