#include "hip/hip_runtime.h"
#include "AddArray.h"

#include <iostream>
#include <assert.h>

#include "Kernel.h"
#include "GM.h"
#include "VectorTools.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addArray(float* ptrGMV1 , float* ptrGMV2 , float* ptrGMW , int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddArray::AddArray(const Grid& grid , float* ptrV1 , float* ptrV2 , float* ptrW , int n) :
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n), //
	dg(grid.dg), //
	db(grid.db)
    {
    this->sizeVector = n*sizeof(float); // TODO addArray // octet


    // MM (malloc Device)
	{
	GM::malloc(&ptrGMV1, sizeVector);
	GM::malloc(&ptrGMV2, sizeVector);
	GM::malloc(&ptrGMW, sizeVector);
	// TODO addArray
	}

    }

AddArray::~AddArray(void)
    {
    //MM (device free)
	{
	GM::free(ptrGMV1);
	GM::free(ptrGMV2);
	GM::free(ptrGMW);


	// TODO addArray
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * override
 */
void AddArray::run()
    {
    // MM (copy Host->Device)
	{
	GM::memcpyHToD(ptrGMV1, ptrV1, sizeVector);
	GM::memcpyHToD(ptrGMV2, ptrV2, sizeVector);

	// TODO addArray
	}

    // TODO addArray // call kernel // assynchrone
	addArray<<<dg,db>>>(ptrGMV1, ptrGMV2, ptrGMW, n);

    //Kernel::synchronize();// inutile

    // MM (Device -> Host)
	{
	// TODO addArray // MM barier de synchronisation implicite
	GM::memcpyDToH(ptrW, ptrGMW, sizeVector);
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
