#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "ReductionAdd.cu.h"

#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(float* tabSM,int nbSlice);
static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void sliceSM(int nbSlice , float* ptrPiHatGM)
    {
    // TODO SliceSM
    extern __shared__ float tabSM[];
    reductionIntraThread(tabSM, nbSlice);

    __syncthreads();
    ReductionAdd::reduce(tabSM, ptrPiHatGM);

    // Reception tabSM
    // ReductionIntraThread
    // Reduction de tabSM (use tools ReductionAdd)

    // __syncthreads(); necessaire? ou?
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * remplit la sm
 */
__device__ void reductionIntraThread(float* tabSM , int nbSlice)
    {
    // TODO SliceSM
	const int TID = Thread2D::tid();
        const int TID_LOCAL = Thread2D::tidLocal();
        const int NB_THREAD = Thread2D::nbThread();
        const float DX = 1 / (float)nbSlice;



        int s = TID;
        float sum = 0;
        float xs;

        while (s < nbSlice)
            {
            xs = s * DX;
            sum += f(xs);
            s += NB_THREAD;
            }

        tabSM[TID_LOCAL] = sum * DX;



    // Warning: Il faut employer TID et TID_LOCAL
    }

__device__ float f(float x)
    {
    // TODO SliceSM
    return 4.f / (1.f + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

