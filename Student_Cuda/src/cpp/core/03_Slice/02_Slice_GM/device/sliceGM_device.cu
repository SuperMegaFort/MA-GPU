#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * <pre>
 * Chaque thread effecteur une reduction intrathread avec le patern d'entrelacement,
 * puis stocke son résultat dans SA case dans tabGM
 *
 * tabGM est un tableau promu, qui a autant de case que de thread
 * </pre>
 */

__global__ void reductionIntraThreadGM(float* tabGM , int nbSlice)
    {
    const int NB_THREAD = Thread2D::nbThread();
    const int TID = Thread2D::tid();
    const float DX = 1 / (float)nbSlice;

    int s = TID;
    float sum = 0;
    float xs;

    while (s < nbSlice)
     {
	 xs = s * DX;
	 sum += f(xs);
	 s += NB_THREAD;
     }
    tabGM[TID] = sum * DX;
    }
/**
 * <pre>
 * Effectue la reduction de tabGM cote device, par ecrasement 2 à 2 successif.
 * Ce kernel d ecrasement est appeler depuis le host dans une boucle, avec le bon nombre de thread
 *
 * Hypothese : |tabGM| est une puissance de 2
 *
 * Output: le resultat de la reduction est tans tabGM[0]
 * </pre>
 */
/*
__global__ void ecrasementGM(float* tabGM , int middle)
    {
    // TODO SliceGM

    int tid = Thread1D::tid();
    if (tid < middle) {
	tabGM[tid] += tabGM[tid + middle];
        }
    }
*/
__global__ void ecrasementGM(float* tabGM , int middle)
    {
    const int TID = Thread2D::tid();
    tabGM[TID] = tabGM[TID] + tabGM[TID + middle];
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    // TODO SliceGM
    return 4.f / (1.f + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

