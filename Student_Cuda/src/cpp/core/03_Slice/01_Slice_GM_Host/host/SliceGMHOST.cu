#include "hip/hip_runtime.h"
#include "SliceGMHOST.h"

#include <iostream>
#include <assert.h>

#include "GM.h"
#include "Maths.h"
#include "Hardware.h"
#include "Kernel.h"

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void reductionIntraThreadGMHOST(float* tabGM,int nbSlice);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

SliceGMHOST::SliceGMHOST(Grid grid , int nbSlice , double* ptrPiHat , bool isVerbose) :
	RunnableGPU(grid, "SliceGM-HOST-" + to_string(nbSlice), isVerbose), // classe parente
//
	nbSlice(nbSlice), //
	ptrPiHat(ptrPiHat) //
    {
    // ntabGM
	{
	this->nTabGM = grid.threadCounts(); // TODO SliceGMHOST // le nombre de case de tabGM. Indication :  grid.threadCounts() donne le nombre de thread ed la grille


	// Warning : Advanced : Et si plus de threads que slices? complique! (pas utile de le coder)
	}

    // MM
	{
	this->sizeTabGM =n*sizeof(float);//  TODO SliceGMHOST // la taille en octet de tabGM [octet]

	}
    }

SliceGMHOST::~SliceGMHOST(void)
    {
    //MM (device free)
	{
	// TODO SliceGMHOST
	GM::free(ptrGMV1);

	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * <pre>
 * Idea globale
 *
 *	Etape 0 : Promotion d'un tableau en GM (MemoryManagement MM)		(Dans le constructeur)
 * 	Etape 1 : Reduction intra-thread dans un tableau promu en GM
 * 	Etape 2 : Copy du tableau coter host
 * 	Etape 3 : Reduction  du tableau coter host
 * 	Etape 4 : Destruction GM						(Dans le destructeur)
 *
 * </pre>
 */
void SliceGMHOST::run()
    {
    // TODO SliceGMHOST // call the kernel
    reductionIntraThreadGMHOST<<<dg,db>>>(float* tabGM,int nbSlice);

    // Indication:
    // 		dg et db sont stokcer dans la classe parente
    // 		vous pouvez les utiliser directement
    // 		exemple : reductionIntraThreadGMHOST<<<dg,db>>>(...)

    reductionHost();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Reduction paralle sur cpu du tableau promu en GM, ramener coter host
 */
void SliceGMHOST::reductionHost()
    {
    // 1) Creer un tableau de bonne dimension (sur la pile, possible ssi petit, sinon sur la tas)
    // 2) Transferer la tabGM dedans
    // 3) Reduction sequentiel cote host
    // 4) finalisation du calcul de ptrPiHat

    // TODO SliceGMHOST
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
