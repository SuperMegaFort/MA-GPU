#include "hip/hip_runtime.h"
#include "Mandelbrot3DMath.h"
#include "Thread2D.cu.h"
#include "Indices.cu.h"

#include "cudas.h"

#include "DomainMath3D_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * w nbPoint en x
 * h nbPoint en y
 */
__global__ void mandelbrot3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , DomainMath3D domaineMath , int n)
    {
    Mandelbrot3DMath mandelbrot3DMath(n);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    const int WH = w * h;

    float x;
    float y;
    float DX;
    float DY;

    int vertexI;
    int vertexJ;

    domaineMath.delta(w, h, &DX, &DY);

    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &vertexI, &vertexJ); // update (vertexI, vertexJ)

	domaineMath.toXY(DX, DY, vertexI, vertexJ, &x, &y);

	mandelbrot3DMath.xyz(&tabVerticesColorGM[s], &tabVerticesXYZGM[s], x, y);

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

