#include "hip/hip_runtime.h"
#include "Mandelbrot3D_RGBA.h"

#include <assert.h>

#include "Maths.h"
#include "cudas.h"

using std::to_string;

using gpu::DomainMath3D;
using gpu::SurfaceStrip_RGBA_uchar4;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot3DCuda(float3* tabVerticesXYZGM, uchar4* tabVerticesColorGM, int w, int h, DomainMath3D domaineMath, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Mandelbrot3D_RGBA::Mandelbrot3D_RGBA(const Grid& grid , int w , int h , const DomainMath3D& domaineMath , int nMin,int nMax) :
	// Parent
	SurfaceStrip_RGBA_uchar4(w, h, domaineMath), //
	// Inputs
	dg(grid.dg), //
	db(grid.db), //
	// Inputs animation
	nMin(nMin), nMax(nMax),
	// Tools animation
	variateurN(Interval<int>(nMin, nMax), 1) // tools animation
    {
    // Tools
    this->title = "Mandelbrot_RGBA_uchar4";
    this->n = nMin;
    }

Mandelbrot3D_RGBA::~Mandelbrot3D_RGBA()
    {
    // rien
    }

/*-------------------------*\
 |*	Surcharge	   *|
 \*------------------------*/

/**
 * Override
 * Call periodicly by the api
 */
void Mandelbrot3D_RGBA::fillVertex(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , uint w , uint h , const gpu::DomainMath3D& domaineMath)
    {
    mandelbrot3DCuda<<<dg,db>>>(tabVerticesXYZGM,tabVerticesColorGM,w,h,domaineMath,n);
    }

/**
 * Override
 * Call periodicly by the api
 */
void Mandelbrot3D_RGBA::animationStep(bool& isNeedUpdate)
    {
    this->n = variateurN.varierAndGet(); // in [nMIn,nMax]
    }

/**
 * Override
 * Call periodicly by the api
 */
void Mandelbrot3D_RGBA::paintPrimitives(Graphic2D& graphic2D)
    {
    graphic2D.setFont(TIMES_ROMAN_24);

    float r = 0.5f;
    float g = 0.5f;
    float b = 0.5f;

    graphic2D.setColorRGB(r, g, b);

    // TOP : Para Animation
	{
	string message = "n = " + to_string(n);
	graphic2D.drawTitleTop(message);
	}

	// BOTTOM : Title
	{
	graphic2D.drawTitleBottom(title);
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

