#include "hip/hip_runtime.h"
#include "Rippling3DMath.h"
#include "Thread2D.cu.h"
#include "Indices.cu.h"

#include "cudas.h"

#include "DomainMath3D_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * w nbPoint en x
 * h nbPoint en y
 *
 * domaineMath pas utiliser ici
 */
__global__ void rippling3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , float t)
    {
    Rippling3DMath rippling3DMath(w, h, t);

    // Indication :
    //		(I1) 	comme en 2D
    //		(I2) 	domaineMath pas utiliser ici

    // TODO
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

