#include "hip/hip_runtime.h"
#include "Rippling3DMath.h"
#include "Thread2D.cu.h"
#include "Indices.cu.h"

#include "cudas.h"

#include "DomainMath3D_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * w nbPoint en x
 * h nbPoint en y
 *
 * domaineMath pas utiliser ici
 */
__global__ void rippling3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , float t)
    {
    Rippling3DMath rippling3DMath(w, h, t);

    // Indication :
    //		(I1) 	comme en 2D
    //		(I2) 	domaineMath pas utiliser ici

    // TODO

//    Rippling3DMath rippling3DMath(n);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    const int WH = w * h;

//    float x;
//    float y;
//    float DX;
//    float DY;
    int i, j;
//    int vertexI;
//    int vertexJ;

//    Rippling3DMath.delta(w, h, &DX, &DY);

    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &i, &j); // update (vertexI, vertexJ)

//	Rippling3DMath.toXY(DX, DY, vertexI, vertexJ, &x, &y);

	rippling3DMath.xyz(&tabVerticesColorGM[s], &tabVerticesXYZGM[s], i, j); // update (sommet s , color s)

	s += NB_THREAD;
	}


    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

