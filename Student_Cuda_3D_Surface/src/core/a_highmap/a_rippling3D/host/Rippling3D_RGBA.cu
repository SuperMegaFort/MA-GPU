#include "hip/hip_runtime.h"
#include "Rippling3D_RGBA.h"

#include <assert.h>

#include "Maths.h"
#include "cudas.h"

using std::to_string;

using gpu::DomainMath3D;
using gpu::SurfaceStrip_RGBA_uchar4;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rippling3DCuda(float3* tabVerticesXYZGM, uchar4* tabVerticesColorGM, int w, int h,float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Rippling3D_RGBA::Rippling3D_RGBA(const Grid& grid , int w , int h , const DomainMath3D& domaineMath , double dt) :
	// Parent
	SurfaceStrip_RGBA_uchar4(w, h, domaineMath), //
	// Inputs
	dg(grid.dg), //
	db(grid.db), //
	// Inputs animation
	dt(dt)
    {
    // Tools
    this->t=0;
    this->title = "Rippling3D_RGBA_uchar4";
    }

Rippling3D_RGBA::~Rippling3D_RGBA()
    {
    // rien
    }

/*-------------------------*\
 |*	Surcharge	   *|
 \*------------------------*/

/**
 * Override
 * Call periodicly by the api
 */
void Rippling3D_RGBA::fillVertex(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , uint w , uint h , const gpu::DomainMath3D& domaineMath)
    {
    rippling3DCuda<<<dg,db>>>(tabVerticesXYZGM,tabVerticesColorGM,w,h,t);
    }

/**
 * Override
 * Call periodicly by the api
 */
void Rippling3D_RGBA::animationStep(bool& isNeedUpdate)
    {
    this->t+=dt;
    }

/**
 * Override
 * Call periodicly by the api
 */
void Rippling3D_RGBA::paintPrimitives(Graphic2D& graphic2D)
    {
    graphic2D.setFont(TIMES_ROMAN_24);

    float r = 0.5f;
    float g = 0.5f;
    float b = 0.5f;

    graphic2D.setColorRGB(r, g, b);

    // TOP : Para Animation
	{
	string message = "t= " + to_string(t);
	graphic2D.drawTitleTop(message);
	}

	// BOTTOM : Title
	{
	graphic2D.drawTitleBottom(title);
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

