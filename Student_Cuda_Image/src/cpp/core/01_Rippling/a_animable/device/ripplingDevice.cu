#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include "Indices.cu.h"

#include "RipplingMath.cu.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

static __device__ void ripplingBaseline(uchar4* tabPixelsGM , uint w , uint h , float t);
static __device__ void ripplingDemi(uchar4* tabPixelsGM , uint w , uint h , float t);
static __device__ void ripplingQuart(uchar4* tabPixelsGM , uint w , uint h , float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void rippling(uchar4* tabPixelsGM , uint w , uint h , float t)
    {
    ripplingBaseline(tabPixelsGM, w, h, t);
   // ripplingDemi(tabPixelsGM, w, h, t);
   // ripplingQuart(tabPixelsGM, w, h, t);
    }

/*----------------------------------------------------------------------*\
 |*			private	 					*|
 \*---------------------------------------------------------------------*/

/**
 * v1
 */
__inline__
__device__ void ripplingBaseline(uchar4* tabPixelsGM , uint w , uint h , float t)
    {
    // TODO instacier RipplingMath

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;

    // TODO Rippling GPU  pattern entrelacement
    }

/**
 * v2 : optimisation
 */
__inline__
__device__ void ripplingDemi(uchar4* tabPixelsGM , uint w , uint h , float t)
    {
    // Indication:
    //		(I1)	Utiliser la symetrie horizontale de l'image
    //		(I2)	Calculer que la demi partie superieur
    //		(I3)	Ranger la couleur calculer dans la demi partie inferieur (effet miroir)
    //			Partez de la fin de l'image, peut-etre

    // TODO Rippling GPU
    }

/**
 * v3 : optimsation : defi (difficile)
 */
__inline__
__device__ void ripplingQuart(uchar4* tabPixelsGM , uint w , uint h , float t)
    {
    // Indication:
    //		(I1)	Utiliser la symetrie horizontale et verticale de l'image
    //		(I2)	Calculer que le quart en huat a gauche
    //		(I3)	Ranger la couleur calculer dans les autres quarts
    //	Warning
    //		(w1)	Necessaire sans doute pour passer le test performance
    //
    //	Contrainte
    //		(C1)	Utiliser toujours le pattern d'entrelacement

    // TODO Rippling GPU
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

