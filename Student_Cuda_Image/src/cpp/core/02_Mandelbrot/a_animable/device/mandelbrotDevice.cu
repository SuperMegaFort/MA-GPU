#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "cudas.h"

#include "real.h"
#include "MandelbrotMath.cu.h"

#include "DomaineMath.h"
#include "Indices.cu.h"

/*----------------------------------------------------------------------*\
 |*			Piste Optimisation 				*|
 \*---------------------------------------------------------------------*/

// optimisation lookup table color
static __device__ void fill(uchar4* tabSM,int n);
static __device__ void color(uchar4* ptrColor,int k, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void mandelbrot(uchar4* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , int n)
    {
    MandelbrotMath mandelbrotMath(n);

    // TODO Mandelbrot GPU :
    //
    // entrelacement
    // s -> (i,j) -> (x,y)
    // appeler colorXY

    double x;
    double y;
    //domaineMath.toXY(i, j, &x, &y); // x et y doivent etre en double! Caster ensuite en real lors du passage à colorXY

    // Probleme : Choix a faire pour le type de (x,y) :
    //
    //			- Travailler en float pour plus de performance?
    //			- Travailler en double pour pouvoir zoomer plus loin dans le fractale?
    //
    // Solution : Travailler avec le typedef real definit dans "real.h"
    //		  Specifier dans "real.h" si vous souhaitez travailler en float ou en double
    //	          Ainsi on passe facilement et proprement d'une version float a une version double
    //
    // Note : domaineMath.toXY travaille lui toujours en double pour vous permettre les deux possibilites ci-dessus
    //
    }

/*----------------------------------------------------------------------*\
 |*			private	 					*|
 \*---------------------------------------------------------------------*/

/**
 * optimisation lookup table color
 */
__inline__
__device__ void fill(uchar4* tabSM,int n)
    {
    // Indications:
    //		(I1)	tabSM a n cases
    //		(I2)	La case k contient les couleurs en RVBA lorsque la suite s est arreter a k
    //		(I3)	Utiliser la methode color ci-dessous pour vous aider
    //		(I4)	Utiliser tabSM dans la partie mandelbrotMath, updater a cet effet quelques prototypes si necessaire
    //			Passer par exemple tabSM au constructeur de mandelbrotMath
    //
    //	Warning
    //		(W1)	Commencer d'abord sans cette piste d'optimisation

    }

/**
 * optimisation lookup table color
 */
__inline__
__device__ void color(uchar4* ptrColor,int k, int n)
    {

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

