#include "hip/hip_runtime.h"
#include <assert.h>

#include "Thread2D.cu.h"
#include "cudas.h"
#include "Indices.cu.h"

#include "Sphere.h"
#include "nbSphere.h"

#include "raytracingCM.cu.h"
#include "RaytracingMath.cu.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void work(uchar4* tabPixelsGM, uint w, uint h, float t, Sphere* tabSpheresDev, int nbSpheres);
static __device__ void copyDevtoSM(Sphere* tabSphereSM , Sphere* tabSphereDev , int nbSpheres);
static __device__ void copyDevtoSM(float* tabSM , float* tabDev , int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres)
    {
    // TODO Raytracing GPU GM

    // Indications :
    //		(I1) Call methode work with good input
    // 		(I2) work contain the algo
    //		(I3) The algo is the same with the GM,CM,SM of TP Ractracing
    }

__global__ void kernelRaytacingSM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres)
    {
    // TODO Raytracing GPU SM

    // Indications :
    //		(I1) Copier les sphere de GM to SM (voir methode de copie en bas)
    // 		(I2) Call work with good input
    //		(I3) Implementer une methode copyGMtoSM
    }

__global__ void kernelRaytacingCM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres)
    {
    // TODO Raytracing GPU CM

    // Indications :
    //		(I1) call work with good input
    // 		(I2) TAB_SPHERES_CM est une variable globale a ce fichier! (Voir le debut de ce fichier, include rayTracingCM.h)
    }

__global__ void kernelRaytacingCM2SM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres)
    {
    // TODO Raytracing GPU SM_BY_CM

    // Indications :
    //		(I1) Copier les sphere de CM to SM (voir methode de copie en bas)
    // 		(I2) Call work with good input
    //		(I3) Implementer une methode copyCMtoSM
    //		     ou : Tip : renommer copyGMtoSM en copyDevToSM et utiliser copyDevToSM avec les bons inputs
    }

/*--------------------------------------*\
 |*		private			*|
 \*-------------------------------------*/

/**
 * Methode commune au 3 kernel ci-dessus.
 * Ici on ne sait pas si derriere tabSpheresDev, c'est
 * 	- de la GM?
 * 	- de la SM?
 * 	- de la CM?
 * Pas d'importance, c'est un pointeur et on travail avec!
 */
__device__ void work(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresDev , int nbSpheres)
    {
    // TODO Raytracing work device side

    // create RaytracingMath
    // entrelacement
    }

/*--------------------------------------*\
 |*		optimisation		*|
 \*-------------------------------------*/

/**
 * tabSphereDev peut etre en GM ou en CM
 * source 	tabSphereDev
 * destination 	tabSphereSM
 */
__device__ void copyDevtoSM(Sphere* tabSphereSM , Sphere* tabSphereDev , int nbSpheres)
    {
    // v1 : copie sphere par sphere
	{
	// TODO en parallel

	// Contraintes:
	//	(C1)	En parallel
	//	(C2)	Tous les threads doivent participer
	}

    // v2 : Optimisation (eviter les banks conflicts) copie de la zone memoire float par float,(ie 4 octets par 4 octets)
//	{
//	float* tabSM = (float*)(void*)tabSphereSM;
//	float* tabDev = (float*)(void*)tabSphereDev;
//	int n = nbSpheres * sizeof(Sphere) / sizeof(float);
//
//	copyDevtoSM(tabSM, tabDev, n); // TODO  coder copyDevtoSM ci-dessous
//	}
    }

/**
 * tabSphereDev peut etre en GM ou en CM
 * source 	tabDev
 * destination 	tabSM
 */
__device__ void copyDevtoSM(float* tabSM , float* tabDev , int n)
    {
    // TODO RaytracingMath

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

