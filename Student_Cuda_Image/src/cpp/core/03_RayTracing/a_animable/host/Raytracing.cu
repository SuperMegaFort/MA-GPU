#include "hip/hip_runtime.h"
#include "Raytracing.h"

#include <iostream>

#include "MM.h"
#include "GM.h"
#include <assert.h>
#include "SphereCreator.h"
#include "nbSphere.h"
#include "Bandwidth.h"
#include "ChronoFactory.h"

using std::cout;
using std::endl;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres);
__global__ void kernelRaytacingSM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres);
__global__ void kernelRaytacingCM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres);
__global__ void kernelRaytacingCM2SM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres);

__host__ void uploadToCM(Sphere* tabSpheres , int nbSpheres);

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

static string titre(const MemoryType& memoryType);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

Raytracing::Raytracing(const Grid& grid , uint w , uint h , float dt , bool isVerbose , int nbSpheres , const MemoryType& memoryType) :
	Animable_I<uchar4>(grid, w, h, titre(memoryType), isVerbose), // super classe
	memoryType(memoryType), //
	dt(dt), //
	nbSpheres(nbSpheres)
    {
    // Tools
    this->sizeSpheres = nbSpheres * sizeof(Sphere);
    this->t = 0;

    // spheres
	{
	Chrono* ptrChrono = ChronoFactory::create();
	SphereCreator sphereCreator(nbSpheres, w, h);
	ptrChrono->stop();
	if (isVerbose)
	    {
	    cout << "\n[SphereCreator] : " << *ptrChrono << endl;
	    }
	delete ptrChrono;

	uploadToDevice(sphereCreator.getTabSphere());
	} // SphereCreator depiler, donc detruit, tabSphere cote host detruit!
    }

Raytracing::~Raytracing()
    {
    switch (memoryType)
	{
    case GM:
	{
	// TODO  Raytracing  GM
	assert(false);	   // to be removed once implemented

	break;
	}
    case CM:
	{
	// Indication:  Rien a detruire pour la CM
	// Note : 	La duree de vie pour la CM est une duree processus.

	// TODO  Raytracing CM
	assert(false);	   // to be removed once implemented

	break;
	}
    case SM:	// ou GM2SM (synonyme)
	{
	// Indication: 	Rien a detruire pour la SM
	// Note :	La duree de vie pour la SM est une duree de vie de kernel, seulement!
	// Warning: 	Par contre si vous avez eu besoin de GM, il faut détruire la GM

	// TODO  Raytracing SM
	assert(false);	   // to be removed once implemented

	break;
	}
    case CM2SM:
	{
	// TODO  Raytracing  CM2SM
	assert(false);	   // to be removed once implemented

	break;
	}
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Raytracing::process(uchar4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    switch (memoryType)
	{
    case GM:
	{
	// Indication : Call the kernel kernelRaytacingGM (prototype line 20, about)

	// TODO Raytracing GM
	assert(false);	   // to be removed once implemented

	break;
	}
    case SM: // ou GM2SM (synonyme)
	{
	// TODO Raytracing SM
	assert(false);	   // to be removed once implemented
	break;
	}
    case CM:
	{
	// TODO Raytracing CM
	assert(false);	   // to be removed once implemented
	break;
	}
    case CM2SM:
	{
	 // TODO Raytracing CM_2_SM
	assert(false);	   // to be removed once implemented
	break;
	}
	}
    }

/**
 * Override
 */
void Raytracing::animationStep()
    {
    t += dt;
    }

/**
 * Override
 */
double Raytracing::getInputGO()
    {
    return NB_SPHERE * sizeof(Sphere) / ((double)1024 * (double)1024 * (double)1024);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

void Raytracing::uploadToDevice(Sphere* tabSpheres)
    {
    Bandwidth bandwidth(sizeSpheres, "\n" + titre(memoryType) + " : Host -> Device :");

    switch (memoryType)
	{
    case GM:
	{
	// But : copier les spheres en GM
	// Indicastions:
	//		(I1) MM pour la GM ( malloc et memcpy)
	// 		(I2) Utiliser la classe GM
	// 		(I3) Regarder bien les attributs de la classe dans le Raytracing.h

	// TODO Raytracing GM uploadToDevice
	assert(false);	   // to be removed once implemented

	break;
	}
    case SM: // ou GM2SM (synonyme)
	{
	// Indications:
	//		(I1) Coter device, on copie GM to SM
	//		(I2) Il faut donc d'abord copier les spheres sur le device!
	//		(I3) Le code est donc le meme que GM

	// TODO Raytracing SM uploadToDevice
	assert(false);	   // to be removed once implemented

	break;
	}
    case CM:
	{
	// But :
	//
	//	Copier les spheres en CM
	//
	// Indications:
	//
	//	(I1) Lisez raytracingCM.cu.h (dans la partie device). Tout est coder.
	//	(I2) Utiliser uploadToCM
	//
	// Notes:
	//
	//	(N1) La CM utilise une variable globale cote device : TAB_SPHERES_CM
	//	(N2) Elle se trouve dans raytracingCM.cu.h (Ligne 12 environ)
	//	(N3) Pour cette raison:
	//			(a) Ce .h est cote device, car ce .h sera inclut cote device dans le fichier du kernel
	//			(b) On a pas acces cote host a TAB_SPHERES_CM
	//			(c) On vous fournit raytracingCM.cu.h qui solutionne tous les problemes
	//	(N4) Le nombre de sphere est defini dans nbSphere.h, qui sera inclut coter host et coter device,
	//	     car on a besoin de connaitre cette quantiter des deux coters


	// TODO Raytracing CM uploadToDevice
	assert(false);	    // to be removed once implemented

	break;
	}
    case CM2SM:
	{
	// Indications:
	//		(I1)Coter device, on copie CM to SM
	//		(I2) Il faut donc d'abord copier les spheres en CM
	//		(I3) Le code est donc le meme que CM

	// TODO uploadToDevice CM2SM ://to be removed once implemented
	assert(false);

	break;
	}
	}

    if (isVerbose)
	{
	cout << endl << bandwidth << endl;
	}
    }

string titre(const MemoryType& memoryType)
    {
    switch (memoryType)
	{
    case GM:
	return "Raytracing-GM-uchar4";

    case SM:
	return "Raytracing-GM2SM-uchar4";

    case CM:
	return "Raytracing-CM-uchar4";

    case CM2SM:
	return "Raytracing-CM2SM-uchar4";

    default:
	{
	assert(false);
	return "ERROR";
	}
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

