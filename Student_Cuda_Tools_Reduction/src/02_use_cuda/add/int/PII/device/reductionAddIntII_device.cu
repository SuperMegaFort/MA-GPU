#include "hip/hip_runtime.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include "ReductionAdd.cu.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/


__global__ void KAddIntProtocoleII(int* ptrSumGM)
    {
    // TODO ReductionAddIntII
    extern __shared__ int tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();

    ReductionAdd::reduce(tabSM, ptrSumGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * TID partout en tabSM
 */
__device__ void reductionIntraThread(int* tabSM)
    {
    // TODO ReductionAddIntII
    const int TID_loc = Thread2D::tidLocal();
    const int TID = Thread2D::tid();

    tabSM[TID_loc] = TID;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

