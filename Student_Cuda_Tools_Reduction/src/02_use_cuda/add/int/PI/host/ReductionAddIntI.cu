#include "hip/hip_runtime.h"
#include "ReductionAddIntI.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KAddIntProtocoleI(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionAddIntI::ReductionAddIntI(const Grid& grid , int* ptrSum , bool isVerbose) :
	RunnableGPU(grid, "ReductionAddIntI-" + to_string(grid.threadCounts()), isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    // TODO ReductionAddIntI
    // MM pour ptrSumGM (oubliez pas initialisation)
    this->sizeSM = sizeof(int)*grid.threadByBlock();

    // Tip:  Il y a une methode dedier pour malloquer un int cote device et l'initialiser a zero
    //
    //		GM::mallocInt0(&ptrSumGM);
    GM::mallocInt0(&ptrSumGM);
    }

ReductionAddIntI::~ReductionAddIntI()
    {
    // TODO ReductionAddIntI
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionAddIntI::run()
    {
    // TODO ReductionAddIntI
    // appeler le kernel
    // recuperer le resulat coter host

    // Tip:  Il y a une methode dedier ramener coter host un int
    //
    //		GM::memcpyDtoH_int(ptrDestination, ptrSourceGM);


    KAddIntProtocoleI<<<grid.dg,grid.db, sizeSM >>>(ptrSumGM);
    GM::memcpyDToH_int(ptrSum, ptrSumGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
