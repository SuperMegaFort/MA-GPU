#include "hip/hip_runtime.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include "ReductionAdd.cu.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/


__global__ void KAddIntProtocoleI(int* ptrSumGM)
    {
    // TODO ReductionAddIntI

    // Reception tabSM
    // ReductionIntraThread
    // ReductionAdd

    // __syncthreads(); // des threads de meme block!// Question : utile? ou?
    extern __shared__ int tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    ReductionAdd::reduce(tabSM, ptrSumGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * 1 partout en tabSM
 */
__device__ void reductionIntraThread(int* tabSM)
    {
    // TODO ReductionAddIntI
    const int TID_loc = Thread2D::tidLocal();
    tabSM[TID_loc] = 1;

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

