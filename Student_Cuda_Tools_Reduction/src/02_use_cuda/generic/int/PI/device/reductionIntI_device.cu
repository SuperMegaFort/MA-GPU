#include "hip/hip_runtime.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include "Reduction.cu.h"


#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM);

// Operateur reduction
static __device__ int add(int x , int y);
static __device__ void addAtomicV1(int* ptrX , int y);
static __device__ void addAtomicV2(int* ptrX , int y);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/


__global__ void KIntProtocoleI(int* ptrSumGM)
    {
    // TODO ReductionIntI
    extern __shared__ int tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    Reduction::reduce(add, addAtomicV1, tabSM, ptrSumGM);
    // nom fonction = pointeur fonction
    //Reduction::reduce(add, addAtomicV2, tabSM, ptrSumGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * 1 partout en tabSM
 */
__device__ void reductionIntraThread(int* tabSM)
    {
    // TODO ReductionIntI
    const int TID_loc = Thread2D::tidLocal();
    tabSM[TID_loc] = 1;
    }

/*----------------------------*\
|*	Operateur reduction    *|
 \*---------------------------*/

__device__ int add(int x , int y)
    {
    // TODO ReductionIntI
    return x+y;
    }

/**
 * Utiliser la methode system
 *
 * 		atomicAdd(pointeurDestination, valeurSource);
 */
__device__ void addAtomicV1(int* ptrX , int y)
    {
    // TODO ReductionIntI
    atomicAdd(ptrX, y);
    }

/**
 * 10x plus lent,mais plus flexible!
 * Necessaire si un vrai operateur atomic n'existe pas
 * Necessaire aussi pour des objets par exemple
 */
__device__ int volatile mutex = 0;	//variable global
__device__ void addAtomicV2(int* ptrX , int y)
    {
    Lock locker(&mutex);
    locker.lock();

    // TODO ReductionIntI
    *ptrX = *ptrX +y; // * = contenu de..
    locker.unlock();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

