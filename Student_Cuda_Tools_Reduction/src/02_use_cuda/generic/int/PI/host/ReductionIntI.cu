#include "hip/hip_runtime.h"
#include "ReductionIntI.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KIntProtocoleI(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionIntI::ReductionIntI(const Grid& grid , int* ptrSum,bool isVerbose) :
	RunnableGPU(grid, "ReductionIntI-" + to_string(grid.threadCounts()),isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    // TODO ReductionIntI
    this->sizeSM = sizeof(int)*grid.threadByBlock();
    GM::mallocInt0(&ptrSumGM);
    }

ReductionIntI::~ReductionIntI()
    {
    // TODO ReductionIntI
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionIntI::run()
    {
    // TODO ReductionIntI
    KIntProtocoleI<<<grid.dg,grid.db, sizeSM >>>(ptrSumGM);
    GM::memcpyDToH_int(ptrSum, ptrSumGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
