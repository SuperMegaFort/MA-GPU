#include "hip/hip_runtime.h"
#include "ReductionIntII.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KIntProtocoleII(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionIntII::ReductionIntII(const Grid& grid , int* ptrSum,bool isVerbose) :
	RunnableGPU(grid, "ReductionIntII-" + to_string(grid.threadCounts()),isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    // TODO ReductionIntII
    this->sizeSM = sizeof(int)*grid.threadByBlock();
    GM::mallocInt0(&ptrSumGM);
    }

ReductionIntII::~ReductionIntII()
    {
    // TODO ReductionIntII
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionIntII::run()
    {
    // TODO ReductionIntII
    KIntProtocoleII<<<grid.dg,grid.db, sizeSM >>>(ptrSumGM);
    GM::memcpyDToH_int(ptrSum, ptrSumGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
