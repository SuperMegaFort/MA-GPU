#include "hip/hip_runtime.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include "Reduction.cu.h"
#include "Lock.cu.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM);

// Operateur reduction
static __device__ int add(int x , int y);
static __device__ void addAtomicV1(int* ptrX , int y);
static __device__ void addAtomicV2(int* ptrX , int y);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void KIntProtocoleII(int* ptrSumGM)
    {
    // TODO ReductionIntII
    extern __shared__ int tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    Reduction::reduce(add, addAtomicV1, tabSM, ptrSumGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * TID partout en tabSM
 */
__device__ void reductionIntraThread(int* tabSM)
    {
    // TODO ReductionIntII
    const int TID_loc = Thread2D::tidLocal();
    const int TID = Thread2D::tid();

    tabSM[TID_loc] = TID;
    }

/*----------------------------*\
|*	Operateur reduction    *|
 \*---------------------------*/

__device__ int add(int x , int y)
    {
    // TODO ReductionIntII
    return x+y;
    }



/**
 * Utiliser la methode system : atomicAdd(pointeurDestination, valeurSource);
 */
__device__ void addAtomicV1(int* ptrX , int y)
    {
    // TODO ReductionIntII
    atomicAdd(ptrX, y);
    }

/**
 * Une alternative, moins performante, mais generalisable serait d'employer un lock
 * Tip : le Lock est implementer avec deux methodes atomic
 */
__device__ int volatile mutex = 0;	//variable global
__device__ void addAtomicV2(int* ptrX , int y)
    {
    Lock locker(&mutex);
    locker.lock();

    // TODO ReductionIntII
    *ptrX = *ptrX +y;
    locker.unlock();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

